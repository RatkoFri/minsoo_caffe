#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include "hyunjin/custom_maths.cuh"
// for generate random number
#include <time.h>
#include "minsoo/fixed.hpp"
#include <iostream>

// global variables
//_MODE_TYPE   mult_type;         // multiplier mode
extern unsigned int mult_type;         // multiplier mode
extern unsigned int drum_k;
/* unsigned int allnumbits;        // #all bits in format  */
/* unsigned int mantissa_numbits;  // #mantissa bits in format */
/* unsigned int fixed_numbits;     // #fixed width in format */
//unsigned int stage1_k;          // #MSBs in mantissa of first stage 
//unsigned int stage2_k;          // #MSBs in mantissa of second stage 
//unsigned int stage3_k;          // #MSBs in mantissa of third stage 
//_RMODE_TYPE  stage1_rmode;      // rounding mode in first stage 
//_RMODE_TYPE  stage2_rmode;      // rounding mode in second stage 
//_RMODE_TYPE  stage3_rmode;      // rounding mode in third stage 
//_RMODE_TYPE  acc_rmode;         // rounding mode after accumulation

//_DMODE_TYPE  data_mode;         // data mode of logarithmic representation
//unsigned int numbitssampling;   // 2 ^ numbitssampling in logarithmic stochastic rounding 
//unsigned int numbits_lsr;       // bits used as weights in logarithmic stochastic rounding 
#define MULT_SWITCH 12 
#define BLOCK_SIZE 32
#define BLOCK_SIZE1 64
#define DRUM_K 4
#define ALLNUMBITS INTBITS+FRACBITS
#include <cstddef> 

namespace caffe {
// Original commented out under this fold
//{{{
template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}
//}}}



template <>
void caffe_gpu_gemm_approx<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
//{{{
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  if (mult_type == 1) // FLOAT
  {
    CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
        N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
    return;
  } 

  // memory for storing results 
  float *dop_A;
  hipMalloc((void **)&dop_A, sizeof(float) * M * K);

  float *dop_B;
  hipMalloc((void **)&dop_B, sizeof(float) * K * N);

  const float alpha_scale = 1;
  const float beta_scale = 0;

  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransA, cuTransA,
      K, M, &alpha_scale, A, lda, &beta_scale, A, lda, dop_A, K));
 
//  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransB, cuTransB,
      N, K, &alpha_scale, B, ldb, &beta_scale, B, ldb, dop_B, N));

  // calling matrix multiplication kernal

  dim3 threadsPerBlock(M, N); // x, y
  dim3 blocksPerGrid(1, 1);
  if (N*M > BLOCK_SIZE1*BLOCK_SIZE1)
  {
    threadsPerBlock.x = BLOCK_SIZE1;
    threadsPerBlock.y = BLOCK_SIZE1;
    blocksPerGrid.x = ceil(double(M)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
  }

  switch (mult_type) 
  {
    case 2: // BFLOAT
     mult_bfloat16<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);    
      break;
    case 3: // ILM1
      //mult_bfloat16_ILM1<<<blocksPerGrid,threadsPerBlock>>>
      mult_bfloat16_ILM1<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 4: // FLOAT
      mult_float<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    default :
      std::cout << "undefined mult_type: " << mult_type << std::endl;
      exit(1);
    break;
  }

  hipDeviceSynchronize();

  // free memory
  hipFree(dop_A);
  hipFree(dop_B);

  return;
}
//}}}


template <>
void caffe_gpu_gemm_approx<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {

    std::cout << "ERROR: caffe_gpu_gemm_approx<double> called" << std::endl;
    throw;
}


template <>
void caffe_gpu_gemv_approx<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
//{{{
  hipblasOperation_t cuTransA =
    (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  if (mult_type == 1) // float
  {
    CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
    return;
  }

  // memory for storing results 
  float *dop_A;
  float *dop_B;
  hipMalloc((void **)&dop_A, sizeof(float) * M * N);

  if (TransA == CblasTrans) 
    hipMalloc((void **) &dop_B, sizeof(float)*M);
  else if (TransA == CblasNoTrans) 
    hipMalloc((void **) &dop_B, sizeof(float)*N);

  //int lda = (TransA == CblasNoTrans) ? M : N;

  const float alpha_scale = 1;
  const float beta_scale = 0;

  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransA, cuTransA,
      M, N, &alpha_scale, A, N, &beta_scale, A, N, dop_A, M));

  if (TransA == CblasTrans) 
  {
    CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), M, x, 1, dop_B, 1));
    CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), M, &alpha_scale, dop_B, 1));
  }
  else if (TransA == CblasNoTrans) 
  {
    CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), N, x, 1, dop_B, 1));
    CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha_scale, dop_B, 1));
  }

  unsigned int row, col;
  if (TransA == CblasTrans) 
  {
    col = 1;
    row = N;
  }
  else if (TransA == CblasNoTrans) 
  {
    col = 1;
    row = M;
  }

  // calling matrix multiplication kernal
  dim3 threadsPerBlock(col, row); // x, y
  dim3 blocksPerGrid(1, 1);
  if (col*row > BLOCK_SIZE*BLOCK_SIZE)
  {
    threadsPerBlock.x = BLOCK_SIZE;
    threadsPerBlock.y = BLOCK_SIZE;
    blocksPerGrid.x = ceil(double(col)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(row)/double(threadsPerBlock.y));
  }
 
  switch (mult_type) 
  {
    case 2: // BFLOAT
      mult_bfloat16<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 3: // ILM1
      mult_bfloat16_ILM1<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 4: // FLOAT
      mult_float<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    default :
      std::cout << "undefined mult_type: " << mult_type << std::endl;
      exit(1);
    break;
  }

  hipDeviceSynchronize();
  // free memory
  hipFree(dop_A);
  hipFree(dop_B);
  
  return;
}

template <>
void caffe_gpu_gemv_approx<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
    
    std::cout << "ERROR: caffe_gpu_gemv_approx<double> called" << std::endl;
    throw;
    
}

template <>
void caffe_gpu_gemm_approxV2<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
//{{{
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  if (mult_type == 1) // FLOAT
  {
    CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
        N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
    return;
  } 

  // memory for storing results 
  float *dop_A;
  hipMalloc((void **)&dop_A, sizeof(float) * M * K);

  float *dop_B;
  hipMalloc((void **)&dop_B, sizeof(float) * K * N);

  const float alpha_scale = 1;
  const float beta_scale = 0;

  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransA, cuTransA,
      K, M, &alpha_scale, A, lda, &beta_scale, A, lda, dop_A, K));
 
//  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransB, cuTransB,
      N, K, &alpha_scale, B, ldb, &beta_scale, B, ldb, dop_B, N));

  // calling matrix multiplication kernal

  dim3 threadsPerBlock(M, N); // x, y
  dim3 blocksPerGrid(1, 1);
  if (N*M > BLOCK_SIZE*BLOCK_SIZE)
  {
    threadsPerBlock.x = BLOCK_SIZE;
    threadsPerBlock.y = BLOCK_SIZE;
    blocksPerGrid.x = ceil(double(M)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
  }

  switch (mult_type) 
  {
    case 2: // BFLOAT
     mult_bfloat16<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);    
      break;
    case 3: // ILM2
      mult_bfloat16_ILM2<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 4: // FLOAT
      mult_float<<<blocksPerGrid,threadsPerBlock>>>
        (dop_B, dop_A, C, N, M, K, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    default :
      std::cout << "undefined mult_type: " << mult_type << std::endl;
      exit(1);
    break;
  }

  hipDeviceSynchronize();

  // free memory
  hipFree(dop_A);
  hipFree(dop_B);

  return;
}
//}}}


template <>
void caffe_gpu_gemm_approxV2<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {

    std::cout << "ERROR: caffe_gpu_gemm_approx<double> called" << std::endl;
    throw;
}



template <>
void caffe_gpu_gemv_approxV2<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
//{{{
  hipblasOperation_t cuTransA =
    (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  if (mult_type == 1) // float
  {
    CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
    return;
  }

  // memory for storing results 
  float *dop_A;
  float *dop_B;
  hipMalloc((void **)&dop_A, sizeof(float) * M * N);

  if (TransA == CblasTrans) 
    hipMalloc((void **) &dop_B, sizeof(float)*M);
  else if (TransA == CblasNoTrans) 
    hipMalloc((void **) &dop_B, sizeof(float)*N);

  //int lda = (TransA == CblasNoTrans) ? M : N;

  const float alpha_scale = 1;
  const float beta_scale = 0;

  // check transposition and scaling 
  CUBLAS_CHECK(hipblasSgeam(Caffe::cublas_handle(), cuTransA, cuTransA,
      M, N, &alpha_scale, A, N, &beta_scale, A, N, dop_A, M));

  if (TransA == CblasTrans) 
  {
    CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), M, x, 1, dop_B, 1));
    CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), M, &alpha_scale, dop_B, 1));
  }
  else if (TransA == CblasNoTrans) 
  {
    CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), N, x, 1, dop_B, 1));
    CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha_scale, dop_B, 1));
  }

  unsigned int row, col;
  if (TransA == CblasTrans) 
  {
    col = 1;
    row = N;
  }
  else if (TransA == CblasNoTrans) 
  {
    col = 1;
    row = M;
  }

  // calling matrix multiplication kernal
  dim3 threadsPerBlock(col, row); // x, y
  dim3 blocksPerGrid(1, 1);
  if (col*row > BLOCK_SIZE*BLOCK_SIZE)
  {
    threadsPerBlock.x = BLOCK_SIZE;
    threadsPerBlock.y = BLOCK_SIZE;
    blocksPerGrid.x = ceil(double(col)/double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(row)/double(threadsPerBlock.y));
  }
 
  switch (mult_type) 
  {
    case 2: // BFLOAT
      mult_bfloat16<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 3: // ILM1
      mult_bfloat16_ILM2<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    case 4: // FLOAT
      mult_float<<<blocksPerGrid,threadsPerBlock>>>
        (dop_A, dop_B, y, row, col, N, DRUM_K,
        ALLNUMBITS, FRACBITS,  alpha, beta);   
      break;
    default :
      std::cout << "undefined mult_type: " << mult_type << std::endl;
      exit(1);
    break;
  }

  hipDeviceSynchronize();
  // free memory
  hipFree(dop_A);
  hipFree(dop_B);
  
  return;
}

//}}}

template <>
void caffe_gpu_gemv_approxV2<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
    
    std::cout << "ERROR: caffe_gpu_gemv_approx<double> called" << std::endl;
    throw;
    
}


//{{{
template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {

  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {

  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

/*
template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}
*/


DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}
//}}}
}  // namespace caffe
